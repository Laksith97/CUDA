
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel function to perform SAXPY (Single-precision A·X Plus Y)
__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) 
        y[i] = a * x[i] + y[i];
}

int main(void) {
    int N = 1 << 20; // Set N to 2^20 (approximately 1 million)
    
    float *x, *y, *d_x, *d_y;

    // Allocate memory on the host (CPU)
    x = (float*) malloc(N * sizeof(float));
    y = (float*) malloc(N * sizeof(float));

    // Allocate memory on the device (GPU)
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    // Initialize host data
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f; // Initialize x with 1.0
        y[i] = 2.0f; // Initialize y with 2.0
    }

    // Copy host data to device
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on N (1M) elements
    saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

    // Copy result back to host
    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    // Check for errors
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 4.0f)); // Expected result: y[i] = 4.0f
    }
    printf("Max error: %f\n", maxError);

    // Free GPU and CPU memory
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    return 0;
}
